#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <random>
#include <vector>
#include <string>
#include <cmath>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>

#include "data_generator.hpp"
#include "group_then_partition.cuh"
#include "groupby_agg_intra_partition.cuh"
#include "CLI11.hpp"

typedef u_int32_t k_type;
typedef u_int32_t v_type;

void write_kv_to_file(const char *filename, k_type *keys, v_type *vals, size_t kv_num)
{
  std::ofstream out(filename);
  for (size_t i = 0; i < kv_num; i++)
  {
    out << keys[i] << "," << vals[i] << "\n";
  }
  out.close();
}

void read_kv_from_file(const char *filename, k_type *keys, v_type *vals, size_t kv_num)
{
  std::ifstream in(filename);
  std::string line;
  size_t i = 0;
  while (std::getline(in, line))
  {
    std::stringstream ss(line);
    std::string key_str;
    std::string val_str;
    std::getline(ss, key_str, ',');
    std::getline(ss, val_str, ',');
    keys[i] = std::stoi(key_str);
    vals[i] = std::stoi(val_str);
    i++;
  }

  in.close();
}

void query_gpu_info() 
{
  int devCount;
  int gpu_ind = 0;
  hipGetDeviceCount(&devCount);
  // std::cout << "devCount:" << devCount << "\n";

  hipDeviceProp_t devProp;
  for (int i = 0; i < devCount; i++)
  {
    hipGetDeviceProperties(&devProp, i);
    // std::cout << "name:" << devProp.name << "\n";
    // std::cout << "major:" << devProp.major << "\n";
  }

  hipSetDevice(gpu_ind);
  // std::cout << "hipSetDevice gpu_ind: " << gpu_ind << "\n\n";
}

__global__ void vectorAdd(float* A, float* B, float* C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

void gpu_warm_up() {
  const int N = 1024;  // Number of elements in arrays
  const int threadsPerBlock = 256;
  const int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

  // Allocate memory on host
  float *h_A = new float[N];
  float *h_B = new float[N];
  float *h_C = new float[N];

  // Initialize input arrays
  for (int i = 0; i < N; ++i) {
    h_A[i] = static_cast<float>(i);
    h_B[i] = std::sqrt(static_cast<float>(i));
  }

  // Allocate memory on device (GPU)
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, N * sizeof(float));
  hipMalloc(&d_B, N * sizeof(float));
  hipMalloc(&d_C, N * sizeof(float));

  // Copy data from host to device
  hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch the kernel
  vectorAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result back to host
  hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

  // Clean up
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
}

int main(int argc, char *argv[])
{
  query_gpu_info();

  CLI::App app;
  // k_type kt_max_version_1 = 0xffffffff;
  k_type kt_max = 0xffffffff;
  v_type vt_max = 0xffffffff;
  size_t kv_num = 1e10;
  size_t cardinality_percentage = 10;
  int dist_kind = 0;
  size_t tile_len = 1e8;
  size_t P = 300;
  size_t nstreams = 8;

  size_t phase1_cpu_worker_num = 12;
  size_t phase2_cpu_worker_num = 0;

  app.add_option("-n", kv_num);
  app.add_option("-c", cardinality_percentage);
  app.add_option("-d", dist_kind);
  app.add_option("-l", tile_len);
  app.add_option("--p1", phase1_cpu_worker_num);
  app.add_option("--p2", phase2_cpu_worker_num);
  CLI11_PARSE(app, argc, argv);

  auto Capacity = tile_len;
  size_t min_load_num = Capacity * 3 / 5;
  size_t max_load_num = Capacity * 4 / 5;

  
  size_t cardinality = kv_num * cardinality_percentage / 100;
  double skew_factor = 0.9;

  k_type *host_keys;
  v_type *host_vals;

  hipHostMalloc(&host_keys, sizeof(k_type) * kv_num);
  hipHostMalloc(&host_vals, sizeof(v_type) * kv_num);

  /// generate kv
  // std::srand(std::time(nullptr));
  // std::random_device r;
  // std::default_random_engine generator(r());
  // std::default_random_engine generator;
  // k_type empty_key = 0xffffffff;
  // generate_various_dist_kv_array<k_type, v_type>(host_keys, host_vals, cardinality, kv_num, skew_factor, generator, dist_kind, empty_key);
  auto start_time = std::chrono::steady_clock::now();
  generate_various_dist_kv_set_multithread_version_2<k_type, v_type>(host_keys,
                                                                     host_vals,
                                                                     cardinality,
                                                                     kv_num,
                                                                     skew_factor,
                                                                     dist_kind,
                                                                     kt_max,
                                                                     vt_max);
  auto end_time = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
  std::cout << "generate_various_dist_kv_set_multithread_version_2 kv_num " <<kv_num << " cardinality " << cardinality << " elapsed time: " << duration.count() << " microseconds\n";
  
  // write_kv_to_file("/data/1e9kv_k4v4_10c.csv", host_keys, host_vals, kv_num);
  // std::cout << "write kv to file done\n";
  ///

  std::vector<par_result> par_result_vec(P);
  gpu_warm_up();
  
  groupby_agg_partition<k_type, v_type>(host_keys,
                                        host_vals,
                                        kv_num,
                                        tile_len,
                                        P,
                                        par_result_vec,
                                        nstreams,
                                        phase1_cpu_worker_num);
  
  
  key_type *host_groupby_keys_result;
  val_type *host_agg_vals_result;
  std::vector<size_t> par_kv_begin;
  std::vector<size_t> par_result_kv_num;

  groupby_agg_intra_partition(par_result_vec,
                              host_groupby_keys_result,
                              host_agg_vals_result,
                              Capacity,
                              min_load_num,
                              max_load_num,
                              nstreams,
                              par_kv_begin,
                              par_result_kv_num,
                              phase2_cpu_worker_num);

  size_t result_kv_num = 0;
  for (size_t i=0; i<par_result_kv_num.size(); i++)
  {
    result_kv_num += par_result_kv_num[i];
  }
  printf("result kv num: %ld\n", result_kv_num);
  
}